#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../Maze/Maze.h"

struct Node
{
    int x, y, h;
};

void dispachAStarCU(unsigned int N, unsigned int M, bool* grid, Elem* solution, Elem* path);

__device__ int heuristic(int row, int col, int goalRow, int goalCol)
{
    return abs(row - goalRow) + abs(col - goalCol);
}

__device__ Node getBestAndErase(Node* list, int size)
{
    int idx = 0;
    Node bestNode = list[idx];
    for (int i = 0; i < size; ++i)
    {
        if (list[i].h < bestNode.h)
        {
            bestNode = list[i];
            idx = i;
        }
    }

    for (int i = idx + 1; i < size; ++i)
    {
        list[i - 1] = list[i];
    }

    return bestNode;
}

__device__ void reverse(Elem* list, int size)
{
    int start = 0;
    int end = size - 1;
    while (start < end)
    {
        Elem temp = list[start];
        list[start] = list[end];
        list[end] = temp;

        ++start;
        --end;
    }
}

__global__ void AStarKernel(unsigned int N, unsigned int M, bool* grid, bool* closed, Node* open, Elem* track, Elem* solution, Elem* path)
{
    int startX = 0;
    int startY = 1;
    int targetX = M - 1;
    int targetY = N - 2;
    int openSize = 0;
    int pathSize = 0;

    Node start{ startX, startY, heuristic(startX, startY, targetX, targetY) };
    open[openSize] = start;
    ++openSize;

    while (openSize > 0)
    {
        Node X = getBestAndErase(open, openSize);
        --openSize;
        path[pathSize] = Elem{ X.x, X.y };
        ++pathSize;

        if (X.x == targetX && X.y == targetY)
        {
            Elem elem{ X.x, X.y };
            int solutionSize = 0;
            while (!(elem.x == startX && elem.y == startY))
            {
                solution[solutionSize] = elem;
                ++solutionSize;
                elem = track[N * elem.y + elem.x];
            }
			solution[solutionSize] = elem;
            ++solutionSize;
            reverse(solution, solutionSize);

            break;
        }

        if (X.x > 0)
        {
            int x = X.x - 1;
            int y = X.y;
            if (grid[N * y + x] == 0 && !closed[N * y + x])
            {
                track[N * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.x < M - 1)
        {
            int x = X.x + 1;
            int y = X.y;
            if (grid[N * y + x] == 0 && !closed[N * y + x])
            {
                track[N * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.y > 0)
        {
            int x = X.x;
            int y = X.y - 1;
            if (grid[N * y + x] == 0 && !closed[N * y + x])
            {
                track[N * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.y < N - 1)
        {
            int x = X.x;
            int y = X.y + 1;
            if (grid[N * y + x] == 0 && !closed[N * y + x])
            {
                track[N * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }

        closed[N * X.y + X.x] = true;
    }
}

void dispachAStarCU(unsigned int N, unsigned int M, bool* grid, Elem* solution, Elem* path)
{
    unsigned int size = N * M;
    bool* dev_grid = 0;
    bool* dev_closed;
    Node* dev_open;
    Elem* dev_track;
    Elem* dev_solution;
    Elem* dev_path;
    hipError_t cudaStatus;

    // Choose which GPU to run on
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (one input, two output)
    cudaStatus = hipMalloc((void**)&dev_grid, size * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_closed, size * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_open, size * sizeof(Node));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_track, size * sizeof(Elem));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_solution, size * sizeof(Elem));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_path, size * sizeof(Elem));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_grid, grid, size * sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    AStarKernel<<<1, 1>>>(N, M, dev_grid, dev_closed, dev_open, dev_track, dev_solution, dev_path);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "AStarKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }

    // Copy solution from GPU buffer to host memory.
    cudaStatus = hipMemcpy(solution, dev_solution, size * sizeof(Elem), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(path, dev_path, size * sizeof(Elem), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_grid);
    hipFree(dev_closed);
    hipFree(dev_open);
    hipFree(dev_track);
    hipFree(dev_solution);
    hipFree(dev_path);
    
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "AStarCU failed!");
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}
