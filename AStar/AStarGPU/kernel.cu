#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../Maze/Maze.h"

struct Node
{
    int x, y, h;
};

struct SetNode
{
    Node node;
    SetNode* left;
    SetNode* right;
};

__device__ SetNode* createNode(Node node)
{
    SetNode* newNode = new SetNode;
    newNode->node = node;
    newNode->left = newNode->right = NULL;
    return newNode;
}

__device__ SetNode* insert(SetNode* root, Node node)
{
    if (root == NULL) {
        return createNode(node);
    }

    if (node.h < root->node.h) {
        root->left = insert(root->left, node);
    } 
    root->right = insert(root->right, node);

    return root;
}

__device__ bool contains(SetNode* root, Node node)
{
    if (root == NULL) {
        return false;
    }

    if (node.x == root->node.x && node.y == root->node.y && node.h == root->node.h) {
        return true;
    }

    if (node.h < root->node.h) {
        return contains(root->left, node);
    } else {
        return contains(root->right, node);
    }
}

__device__ void deleteSet(SetNode* root)
{
    if (root != NULL) {
        deleteSet(root->left);
        deleteSet(root->right);
        delete root;
    }
}

__device__ int heuristic(int row, int col, int goalRow, int goalCol)
{
    return abs(row - goalRow) + abs(col - goalCol);
}

__device__ Node getBestAndErase(Node* list, int size)
{
    int idx = 0;
    Node bestNode = list[idx];
    for (int i = 0; i < size; ++i)
    {
        if (list[i].h < bestNode.h)
        {
            bestNode = list[i];
            idx = i;
        }
    }

    for (int i = idx + 1; i < size; ++i)
    {
        list[i - 1] = list[i];
    }

    return bestNode;
}

__device__ void reverse(Elem* list, int size)
{
    int start = 0;
    int end = size - 1;
    while (start < end)
    {
        Elem temp = list[start];
        list[start] = list[end];
        list[end] = temp;

        ++start;
        --end;
    }
}

__global__ void AStarKernel(unsigned int N, unsigned int M, bool* grid, bool* closed, Node* open, Elem* track, Elem* solution, Elem* path)
{
    int startX = 0;
    int startY = 1;
    int targetX = M - 1;
    int targetY = N - 2;
    int openSize = 0;
    int pathSize = 0;

    Node start{ startX, startY, heuristic(startX, startY, targetX, targetY) };
    open[openSize] = start;
    ++openSize;

    bool found = false;
    while (openSize > 0)
    {
        Node X = getBestAndErase(open, openSize);
        --openSize;
        path[pathSize] = Elem{ X.x, X.y };
        ++pathSize;

        if (X.x == targetX && X.y == targetY)
        {
            found = true;
            Elem elem{ X.x, X.y };
            int solutionSize = 0;
            while (!(elem.x == startX && elem.y == startY))
            {
                solution[solutionSize] = elem;
                ++solutionSize;
                elem = track[M * elem.y + elem.x];
            }
			solution[solutionSize] = elem;
            ++solutionSize;
            reverse(solution, solutionSize);

            break;
        }

        if (X.x > 0)
        {
            int x = X.x - 1;
            int y = X.y;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.x < M - 1)
        {
            int x = X.x + 1;
            int y = X.y;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.y > 0)
        {
            int x = X.x;
            int y = X.y - 1;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.y < N - 1)
        {
            int x = X.x;
            int y = X.y + 1;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }

        closed[M * X.y + X.x] = true;
    }
    
    if (!found)
        printf("No solution found\n");
}

struct AStarArgs
{
    unsigned int N;
    unsigned int M;
    bool* brid;
    bool* closed;
    Node* open;
    Elem* track;
    Elem* solution;
    Elem* path;
};

void dispachAStarCU(unsigned int N, unsigned int M, bool* grid, Elem* solution, Elem* path)
{
    unsigned int size = N * M;
    bool* dev_grid = 0;
    bool* dev_closed;
    Node* dev_open;
    Elem* dev_track;
    Elem* dev_solution;
    Elem* dev_path;
    hipError_t cudaStatus;

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Choose which GPU to run on
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (one input, two output)
    cudaStatus = hipMallocAsync((void**)&dev_grid, size * sizeof(bool), stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocAsync((void**)&dev_closed, size * sizeof(bool), stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocAsync((void**)&dev_open, size * sizeof(Node), stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocAsync((void**)&dev_track, size * sizeof(Elem), stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocAsync((void**)&dev_solution, size * sizeof(Elem), stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocAsync((void**)&dev_path, size * sizeof(Elem), stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpyAsync(dev_grid, grid, size * sizeof(bool), hipMemcpyHostToDevice, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    AStarKernel<<<1, 1, 0, stream>>>(N, M, dev_grid, dev_closed, dev_open, dev_track, dev_solution, dev_path);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "AStarKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }

    // Copy solution from GPU buffer to host memory.
    cudaStatus = hipMemcpyAsync(solution, dev_solution, size * sizeof(Elem), hipMemcpyDeviceToHost, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpyAsync(path, dev_path, size * sizeof(Elem), hipMemcpyDeviceToHost, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFreeAsync(dev_grid, stream);
    hipFreeAsync(dev_closed, stream);
    hipFreeAsync(dev_open, stream);
    hipFreeAsync(dev_track, stream);
    hipFreeAsync(dev_solution, stream);
    hipFreeAsync(dev_path, stream);

    hipStreamDestroy(stream);
    
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "AStarCU failed!");
    }
}
