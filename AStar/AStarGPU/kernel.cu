#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "../Maze/Maze.h"

struct Node
{
    int x, y, h;
};

__device__ int heuristic(int row, int col, int goalRow, int goalCol)
{
    return abs(row - goalRow) + abs(col - goalCol);
}

__device__ Node getBestAndErase(Node* list, int size)
{
    int idx = 0;
    Node bestNode = list[idx];
    for (int i = 0; i < size; ++i)
    {
        if (list[i].h < bestNode.h)
        {
            bestNode = list[i];
            idx = i;
        }
    }

    for (int i = idx + 1; i < size; ++i)
    {
        list[i - 1] = list[i];
    }

    return bestNode;
}

__device__ void reverse(Elem* list, int size)
{
    int start = 0;
    int end = size - 1;
    while (start < end)
    {
        Elem temp = list[start];
        list[start] = list[end];
        list[end] = temp;

        ++start;
        --end;
    }
}

struct AStarArgs
{
    unsigned int N;
    unsigned int M;
    bool* grid;
    bool* closed;
    Node* open;
    Elem* track;
    Elem* solution;
    Elem* path;
};

__global__ void AStarKernel(AStarArgs* args, int argsSize)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= argsSize)
        return;

    unsigned int N = args[tid].N;
    unsigned int M = args[tid].M;
    bool* grid = args[tid].grid;
    bool* closed = args[tid].closed;
    Node* open = args[tid].open;
    Elem* track = args[tid].track;
    Elem* solution = args[tid].solution;
    Elem* path = args[tid].path;

    int startX = 0;
    int startY = 1;
    int targetX = M - 1;
    int targetY = N - 2;
    int openSize = 0;
    int pathSize = 0;

    Node start{ startX, startY, heuristic(startX, startY, targetX, targetY) };
    open[openSize] = start;
    ++openSize;

    while (openSize > 0)
    {
        Node X = getBestAndErase(open, openSize);
        --openSize;
        path[pathSize] = Elem{ X.x, X.y };
        ++pathSize;

        if (X.x == targetX && X.y == targetY)
        {
            Elem elem{ X.x, X.y };
            int solutionSize = 0;
            while (!(elem.x == startX && elem.y == startY))
            {
                solution[solutionSize] = elem;
                ++solutionSize;
                elem = track[M * elem.y + elem.x];
            }
			solution[solutionSize] = elem;
            ++solutionSize;
            reverse(solution, solutionSize);

            break;
        }

        if (X.x > 0)
        {
            int x = X.x - 1;
            int y = X.y;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.x < M - 1)
        {
            int x = X.x + 1;
            int y = X.y;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.y > 0)
        {
            int x = X.x;
            int y = X.y - 1;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }
        if (X.y < N - 1)
        {
            int x = X.x;
            int y = X.y + 1;
            if (grid[M * y + x] == 0 && !closed[M * y + x])
            {
                track[M * y + x] = Elem{ X.x, X.y };
                open[openSize] = Node{ x, y, heuristic(x, y, targetX, targetY) };
                ++openSize;
            }
        }

        closed[M * X.y + X.x] = true;
    }
}


void dispachAStarCU(const std::vector<const Grid*>& grids, std::vector<std::vector<Elem>>& paths, std::vector<std::vector<Elem>>& solutions)
{
    bool* dev_grid;
    bool* dev_closed;
    Node* dev_open;
    Elem* dev_track;
    Elem* dev_solution;
    Elem* dev_path;
    AStarArgs* dev_args;

    hipError_t cudaStatus;
    unsigned int offset = 0;
    std::vector<AStarArgs> args;

    int numOfThreads = 1;
    int numOfBlocks = 1;

    if (grids.size() > 32)
    {
        numOfThreads = 32;
        numOfBlocks = std::ceil((double)grids.size() / (double)numOfThreads);
    }
    else
    {
        numOfThreads = grids.size();
    }
    
    unsigned int size = 0;
    for (const Grid* grid : grids)
    {
        unsigned int N = grid->size();
        unsigned int M = grid->at(0).size();
		size += N * M;
    }


    cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
		
	cudaStatus = hipMalloc((void**)&dev_grid, size * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_closed, size * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_open, size * sizeof(Node));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_track, size * sizeof(Elem));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_solution, size * sizeof(Elem));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_path, size * sizeof(Elem));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    for (const Grid* grid : grids)
    {
        unsigned int N = grid->size();
        unsigned int M = grid->at(0).size();
		unsigned int size = N * M;

        AStarArgs arg;
        arg.N = N;
        arg.M = M;
        arg.grid = dev_grid + offset;
        arg.closed = dev_closed + offset;
        arg.open = dev_open + offset;
        arg.track = dev_track + offset;
        arg.solution = dev_solution + offset;
        arg.path = dev_path + offset;

        bool* gridCpy = new bool[size];
        for (size_t i = 0; i < N; ++i)
        {
            for (size_t j = 0; j < M; ++j)
            {
                gridCpy[M * i + j] = grid->at(i)[j];
            }
        }

		cudaStatus = hipMemcpy(arg.grid, gridCpy, size * sizeof(bool), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

        args.push_back(arg);
        delete[] gridCpy;
        offset += size;
    }
    
    cudaStatus = hipMalloc((void**)&dev_args, args.size() * sizeof(AStarArgs));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_args, args.data(), args.size() * sizeof(AStarArgs), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
    AStarKernel<<<numOfBlocks, numOfThreads>>>(dev_args, args.size());

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "AStarKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }

    for (size_t i = 0; i < args.size(); ++i)
    {
		// Copy solution from GPU buffer to host memory.
        unsigned int size = args[i].N * args[i].M;
		cudaStatus = hipMemcpyAsync(solutions[i].data(), args[i].solution, size * sizeof(Elem), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpyAsync(paths[i].data(), args[i].path, size * sizeof(Elem), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
    }

Error:
    hipFree(dev_grid);
    hipFree(dev_closed);
    hipFree(dev_open);
    hipFree(dev_track);
    hipFree(dev_solution);
    hipFree(dev_path);
    hipFree(dev_args);
    
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "AStarCU failed!");
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}
